#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date April 2022

       @author Azzam Haidar
       @author Tingxing Dong
       @author Ahmad Abdelfattah

       @generated from magmablas/zgetf2_kernels.cu, normal z -> s, Wed Apr 20 17:38:06 2022
*/

#include "magma_internal.h"
#include "batched_kernel_param.h"
#include "magma_templates.h"
#include "shuffle.cuh"
#include "sgetf2_devicefunc.cuh"

#define PRECISION_s

#define A(i, j)  (A + (i) + (j)*lda)   // A(i, j) means at i row, j column

/******************************************************************************/
__global__ void
isamax_kernel_batched(
        int length, float **x_array, int xi, int xj, int lda, int incx,
        magma_int_t** ipiv_array, int ipiv_i,
        magma_int_t *info_array, int step, int gbstep)
{
    extern __shared__ float sdata[];
    const int batchid = blockIdx.x;

    int tx = threadIdx.x;
    const float *x = x_array[batchid] + xj * lda + xi;
    magma_int_t *ipiv           = ipiv_array[batchid] + ipiv_i;

    float *shared_x = sdata;
    int *shared_idx = (int*)(shared_x + zamax);

    isamax_devfunc(length, x, incx, shared_x, shared_idx);

    if (tx == 0) {
        *ipiv  = shared_idx[0] + step + 1; // Fortran Indexing & adjust ipiv
        if (shared_x[0] == MAGMA_D_ZERO) {
            info_array[batchid] = shared_idx[0] + step + gbstep + 1;
        }
    }
}


/******************************************************************************/
__global__ void
isamax_kernel_native(
        int length, magmaFloat_ptr x, int incx,
        magma_int_t* ipiv, magma_int_t *info,
        int step, int gbstep)
{
    extern __shared__ float sdata[];
    const int tx = threadIdx.x;

    float *shared_x = sdata;
    int *shared_idx = (int*)(shared_x + zamax);

    isamax_devfunc(length, x, incx, shared_x, shared_idx);
    if (tx == 0) {
        *ipiv  = shared_idx[0] + step + 1; // Fortran Indexing
        if (shared_x[0] == MAGMA_D_ZERO) {
            (*info) = shared_idx[0] + step + gbstep + 1;
        }
    }
}


/***************************************************************************//**
    Purpose
    -------

    ISAMAX find the index of max absolute value of elements in x and store the index in ipiv

    This is an internal routine that might have many assumption.

    Arguments
    ---------

    @param[in]
    length       INTEGER
            On entry, length specifies the size of vector x. length >= 0.


    @param[in]
    x_array     Array of pointers, dimension (batchCount).
            Each is a REAL array of dimension


    @param[in]
    xi      INTEGER
            Row offset, internal use

    @param[in]
    xj      INTEGER
            Column offset, internal use

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    step    INTEGER
            the offset of ipiv

    @param[in]
    lda    INTEGER
            The leading dimension of each array A, internal use to find the starting position of x.

    @param[out]
    ipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    gbstep    INTEGER
            the offset of info, internal use

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_iamax_batched
*******************************************************************************/
extern "C" magma_int_t
magma_isamax_batched(
        magma_int_t length,
        float **x_array, magma_int_t xi, magma_int_t xj, magma_int_t lda, magma_int_t incx,
        magma_int_t** ipiv_array, magma_int_t ipiv_i,
        magma_int_t step, magma_int_t gbstep, magma_int_t *info_array,
        magma_int_t batchCount, magma_queue_t queue)
{
    if (length == 0 ) return 0;

    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    int chunk = magma_ceildiv( length, zamax );

    isamax_kernel_batched<<< grid, threads, zamax * (sizeof(float) + sizeof(int)), queue->hip_stream() >>>
    (length, x_array, xi, xj, lda, incx, ipiv_array, ipiv_i, info_array, step, gbstep);

    return 0;
}


/******************************************************************************/
// For use in magma_isamax_native only
// hipblasIsamax always writes 32bit pivots, so make sure it is magma_int_t
__global__ void magma_spivcast(magma_int_t* dipiv)
{
    // uses only 1 thread
    int* address = (int*)dipiv;
    int pivot = *address;          // read the value written by cuBLAS (int)
    *dipiv = (magma_int_t)pivot;    // write it back in the same address as dipiv
}

/******************************************************************************/
extern "C" magma_int_t
magma_isamax_native(
    magma_int_t length,
    magmaFloat_ptr x, magma_int_t incx,
    magma_int_t* ipiv, magma_int_t *info,
    magma_int_t step, magma_int_t gbstep, magma_queue_t queue)
{
    if (length == 0 ) return 0;

    // TODO: decide the best isamax for all precisions
    if( length <= 15360 ) {
        dim3 grid(1, 1, 1);
        dim3 threads(zamax, 1, 1);

        isamax_kernel_native<<< grid, threads, zamax * (sizeof(float) + sizeof(int)), queue->hip_stream() >>>
        (length, x, incx, ipiv, info, step, gbstep);
    }
    else {
    #ifdef MAGMA_HAVE_CUDA
        hipblasPointerMode_t ptr_mode;
        hipblasGetPointerMode(queue->cublas_handle(), &ptr_mode);
        hipblasSetPointerMode(queue->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE);

        hipblasIsamax(queue->cublas_handle(), length, x, 1, (int*)(ipiv));
        magma_spivcast<<< 1, 1, 0, queue->hip_stream() >>>( ipiv );

        hipblasSetPointerMode(queue->cublas_handle(), ptr_mode);
    #elif defined(MAGMA_HAVE_HIP)
        hipblasPointerMode_t ptr_mode;
        hipblasGetPointerMode(queue->hipblas_handle(), &ptr_mode);
        hipblasSetPointerMode(queue->hipblas_handle(), HIPBLAS_POINTER_MODE_DEVICE);

        hipblasIsamax(queue->hipblas_handle(), length, (const float*)x, 1, (int*)(ipiv));
        magma_spivcast<<< 1, 1, 0, queue->hip_stream() >>>( ipiv );

        hipblasSetPointerMode(queue->hipblas_handle(), ptr_mode);
    #endif

        adjust_ipiv( ipiv, 1, step, queue);
    }
    return 0;
}

/******************************************************************************/
__global__
void sswap_kernel_batched(
        magma_int_t n,
        float **x_array, magma_int_t xi, magma_int_t xj, magma_int_t incx,
        magma_int_t step, magma_int_t** ipiv_array)
{
    const int batchid = blockIdx.x;
    float *x = x_array[batchid] + xj * incx + xi;
    magma_int_t *ipiv = ipiv_array[batchid] + xi;

    sswap_device(n, x, incx, step, ipiv);
}


/******************************************************************************/
__global__
void sswap_kernel_native( magma_int_t n,
                          magmaFloat_ptr x, magma_int_t incx,
                          magma_int_t step, magma_int_t* ipiv)
{
    sswap_device(n, x, incx, step, ipiv);
}


/***************************************************************************//**
    Purpose
    -------

    sswap two row in x.  index (ipiv[step]-1)-th and index step -th

    This is an internal routine that might have many assumption.

    Arguments
    ---------

    @param[in]
    n       INTEGER
            On entry, n specifies the size of vector x. n >= 0.


    @param[in]
    dA_array  Array of pointers, dimension (batchCount).
            Each is a REAL array of dimension


    @param[in]
    ai      INTEGER
            Row offset, internal use.

    @param[in]
    aj      INTEGER
            Column offset, internal use.

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    step    INTEGER
            The starting address of matrix C in A.  LDDA >= max(1,M).

    @param[out]
    ipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).


    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_swap_batched
*******************************************************************************/
extern "C" magma_int_t
magma_sswap_batched( magma_int_t n,
                     float **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t incx,
                     magma_int_t step, magma_int_t** ipiv_array,
                     magma_int_t batchCount, magma_queue_t queue)
{
    /*
    sswap two row: (ipiv[step]-1)th and step th
    */
    if ( n  > MAX_NTHREADS)
    {
        fprintf( stderr, "%s nb=%lld > %lld, not supported\n",
                 __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }
    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    sswap_kernel_batched
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, dA_array, ai, aj, incx, step, ipiv_array);
    return 0;
}


/******************************************************************************/
extern "C" void
magma_sswap_native( magma_int_t n, magmaFloat_ptr x, magma_int_t incx,
                    magma_int_t step, magma_int_t* ipiv,
                    magma_queue_t queue)
{
    /*
    sswap two row: (ipiv[step]-1)th and step th
    */
    if ( n  > MAX_NTHREADS){
        fprintf( stderr, "%s nb=%lld > %lld, not supported\n",
                 __func__, (long long) n, (long long) MAX_NTHREADS );
    }
    dim3 grid(1, 1, 1);
    dim3 threads(zamax, 1, 1);

    sswap_kernel_native
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, x, incx, step, ipiv);
}

/******************************************************************************/
template<int N>
__global__
void sscal_sger_1d_kernel_native( int m,
                                magmaFloat_ptr dA, int lda,
                                magma_int_t *info, int step, int gbstep)
{
    // This dev function has a return statement inside, be sure
    // not to merge it with another dev function. Otherwise, the
    // return statement should be converted into an if-statement
    sscal_sger_device<N>(m, dA, lda, info, step, gbstep);
}


/******************************************************************************/
__global__
void sscal_sger_1d_generic_kernel_native( int m, int n,
                                magmaFloat_ptr dA, int lda,
                                magma_int_t *info, int step, int gbstep)
{
    // This dev function has a return statement inside, be sure
    // not to merge it with another dev function. Otherwise, the
    // return statement should be converted into an if-statement
    sscal_sger_generic_device(m, n, dA, lda, info, step, gbstep);
}


/******************************************************************************/
template<int N>
__global__
void sscal_sger_1d_kernel_batched(int m, float **dA_array, int ai, int aj, int lda, magma_int_t *info_array, int step, int gbstep)
{
    const int batchid = blockIdx.z;
    float* dA = dA_array[batchid] + aj * lda + ai;
    magma_int_t *info = &info_array[batchid];
    sscal_sger_device<N>(m, dA, lda, info, step, gbstep);
}


/******************************************************************************/
__global__
void sscal_sger_1d_generic_kernel_batched(int m, int n, float **dA_array, int ai, int aj, int lda, magma_int_t *info_array, int step, int gbstep)
{
    const int batchid = blockIdx.z;
    float* dA = dA_array[batchid] + aj * lda + ai;
    magma_int_t *info = &info_array[batchid];
    sscal_sger_generic_device(m, n, dA, lda, info, step, gbstep);
}


/******************************************************************************/
extern "C"
magma_int_t
magma_sscal_sger_batched(
    magma_int_t m, magma_int_t n,
    float **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t lda,
    magma_int_t *info_array, magma_int_t step, magma_int_t gbstep,
    magma_int_t batchCount, magma_queue_t queue)
{
    /*
    Specialized kernel which merged sscal and sger the two kernels
    1) sscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a sger Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( n == 0) return 0;
    if ( n > MAX_NTHREADS ) {
        fprintf( stderr, "%s nb=%lld, > %lld, not supported\n", __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }

    magma_int_t max_batchCount = queue->get_maxBatch();
    const int tbx = 256;
    dim3 threads(tbx, 1, 1);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid(magma_ceildiv(m,tbx), 1, ibatch);

        switch(n){
            case  1: sscal_sger_1d_kernel_batched< 1><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  2: sscal_sger_1d_kernel_batched< 2><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  3: sscal_sger_1d_kernel_batched< 3><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  4: sscal_sger_1d_kernel_batched< 4><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  5: sscal_sger_1d_kernel_batched< 5><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  6: sscal_sger_1d_kernel_batched< 6><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  7: sscal_sger_1d_kernel_batched< 7><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            case  8: sscal_sger_1d_kernel_batched< 8><<<grid, threads, 0, queue->hip_stream()>>>( m, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);break;
            default: sscal_sger_1d_generic_kernel_batched<<<grid, threads, 0, queue->hip_stream()>>>(m, n, dA_array+i, ai, aj, lda, info_array+i, step, gbstep);
        }
    }
    return 0;
}


/******************************************************************************/
extern "C"
magma_int_t
magma_sscal_sger_native(
    magma_int_t m, magma_int_t n,
    magmaFloat_ptr dA, magma_int_t lda,
    magma_int_t *info, magma_int_t step, magma_int_t gbstep,
    magma_queue_t queue)
{
    /*
    Specialized kernel which merged sscal and sger the two kernels
    1) sscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a sger Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( n == 0) return 0;
    if ( n > MAX_NTHREADS ) {
        fprintf( stderr, "%s nb=%lld, > %lld, not supported\n", __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }
    const int tbx = 256;
    dim3 grid(magma_ceildiv(m,tbx), 1, 1);
    dim3 threads(tbx, 1, 1);
    switch(n){
        case 1: sscal_sger_1d_kernel_native<1><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 2: sscal_sger_1d_kernel_native<2><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 3: sscal_sger_1d_kernel_native<3><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 4: sscal_sger_1d_kernel_native<4><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 5: sscal_sger_1d_kernel_native<5><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 6: sscal_sger_1d_kernel_native<6><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 7: sscal_sger_1d_kernel_native<7><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        case 8: sscal_sger_1d_kernel_native<8><<<grid, threads, 0, queue->hip_stream()>>>( m, dA, lda, info, step, gbstep);break;
        default: sscal_sger_1d_generic_kernel_native<<<grid, threads, 0, queue->hip_stream()>>>( m, n, dA, lda, info, step, gbstep);
    }
    return 0;
}


/******************************************************************************/
__global__
void sgetf2trsm_kernel_batched(int ib, int n, float **dA_array, int step, int lda)
{

    extern __shared__ float shared_data[];

    /*
        this kernel does the safe nonblocked TRSM operation
        B = A^-1 * B
    */
    const int batchid = blockIdx.x;

    float *A_start = dA_array[batchid];
    float *A = &(A_start[step + step * lda]);
    float *B = &(A_start[step + (step+ib) * lda]);
    float *shared_a = shared_data;
    float *shared_b = shared_data+ib*ib;

    int tid = threadIdx.x;
    int i,d;


    // Read A and B at the same time to the shared memory (shared_a shared_b)
    // note that shared_b = shared_a+ib*ib so its contiguous
    // I can make it in one loop reading
    if ( tid < ib) {
        #pragma unroll
        for (i=0; i < n+ib; i++) {
            shared_a[tid + i*ib] = A[tid + i*lda];
        }
    }
    __syncthreads();

    if (tid < n) {
        #pragma unroll
        for (d=0;  d < ib-1; d++) {
            for (i=d+1; i < ib; i++) {
                shared_b[i+tid*ib] += (MAGMA_S_NEG_ONE) * shared_a[i+d*ib] * shared_b[d+tid*ib];
            }
        }
    }
    __syncthreads();

    // write back B
    if ( tid < ib) {
        #pragma unroll
        for (i=0; i < n; i++) {
            B[tid + i*lda] = shared_b[tid + i*ib];
        }
    }
}


/***************************************************************************//**
    Purpose
    -------

    sgetf2trsm solves one of the matrix equations on gpu

     B = C^-1 * B

    where C, B are part of the matrix A in dA_array,

    This version load C, B into shared memory and solve it
    and copy back to GPU device memory.
    This is an internal routine that might have many assumption.

    Arguments
    ---------
    @param[in]
    ib       INTEGER
            The number of rows/columns of each matrix C, and rows of B.  ib >= 0.

    @param[in]
    n       INTEGER
            The number of columns of each matrix B.  n >= 0.

    @param[in,out]
    dA_array    Array of pointers, dimension (batchCount).
            Each is a REAL array on the GPU, dimension (LDDA,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ldda    INTEGER
            The leading dimension of each array A.  LDDA >= max(1,M).

    @param[in]
    step    INTEGER
            The starting address of matrix C in A.  LDDA >= max(1,M).

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getf2_batched
*******************************************************************************/
extern "C" void
magma_sgetf2trsm_batched(magma_int_t ib, magma_int_t n, float **dA_array,
                         magma_int_t step, magma_int_t ldda,
                         magma_int_t batchCount, magma_queue_t queue)
{
    if ( n == 0 || ib == 0 ) return;
    size_t shared_size = sizeof(float)*(ib*(ib+n));

    // TODO TODO TODO
    if ( shared_size > (MAX_SHARED_ALLOWED*1024) ) // limit the shared memory to 46K leaving 2K for extra
    {
        fprintf( stderr, "%s: error out of shared memory\n", __func__ );
        return;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(n,ib), 1, 1);

    sgetf2trsm_kernel_batched
    <<< grid, threads, shared_size, queue->hip_stream() >>>
    (ib, n, dA_array, step, ldda);
}


/******************************************************************************/
template<int NB>
__global__ void
sgetf2trsm_2d_kernel( int m, int n,
                           magmaFloat_ptr dA, int ldda,
                           magmaFloat_ptr dB, int lddb)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float sA[NB * NB];
    __shared__ float sB[NB * NB];

    // init sA & sB
    sA[ ty * NB + tx ] = MAGMA_S_ZERO;
    sB[ ty * NB + tx ] = MAGMA_S_ZERO;

    const int nblocks = magma_ceildiv(n, NB);
    const int n_ = n - (nblocks-1) * NB;

    // load A
    if( ty < m && tx < m && tx > ty){
        sA[ty * NB + tx] = dA[ty * ldda + tx];
    }

    if( ty == tx ){
        // ignore diagonal elements
        sA[tx * NB + tx] = MAGMA_S_ONE;
    }
    __syncthreads();

    #pragma  unroll
    for(int s = 0; s < nblocks-1; s++){
        // load B
        if( tx < m ){
            sB[ ty * NB + tx] = dB[ ty * lddb + tx ];
        }

        // no need to sync because each thread column is less than 32
        // solve
        #pragma unroll
        for(int i = 0; i < NB; i++){
            if(tx >  i){
                 sB[ ty * NB + tx ] -= sA[ i * NB + tx ] * sB[ ty * NB + i ];
            }
        }

        // write B
        if( tx < m){
            dB[ ty * lddb + tx ] = sB[ ty * NB + tx ];
        }
        dB += NB * lddb;
    }

    // last, possible partial, block
    if( ty < n_ && tx < m){
        sB[ ty * NB + tx] = dB[ ty * lddb + tx ];
    }

    #pragma unroll
    for(int i = 0; i < NB; i++){
        if(tx >  i){
             sB[ ty * NB + tx ] -= sA[ i * NB + tx ] * sB[ ty * NB + i ];
        }
    }

    if( ty < n_ && tx < m){
        dB[ ty * lddb + tx ] = sB[ ty * NB + tx ];
    }
}


/******************************************************************************/
extern"C" void
magma_sgetf2trsm_2d_native(
    magma_int_t m, magma_int_t n,
    magmaFloat_ptr dA, magma_int_t ldda,
    magmaFloat_ptr dB, magma_int_t lddb,
    magma_queue_t queue)
{
    if( m > 32 ){
        magma_strsm( MagmaLeft, MagmaLower, MagmaNoTrans, MagmaUnit,
                     m, n, MAGMA_S_ONE,
                     dA, ldda,
                     dB, lddb, queue );
        return;
    }

    const int m8 = magma_roundup(m, 8);
    dim3 grid(1, 1, 1);
    dim3 threads(m8, m8, 1);

    switch(m8){
        case  8: sgetf2trsm_2d_kernel< 8><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 16: sgetf2trsm_2d_kernel<16><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 24: sgetf2trsm_2d_kernel<24><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 32: sgetf2trsm_2d_kernel<32><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        default:;
    }
}

/******************************************************************************/
__global__ void
zcomputecolumn_kernel_shared_batched( int m, int paneloffset, int step,
                                      float **dA_array, int ai, int aj,
                                      int lda, magma_int_t **ipiv_array, magma_int_t *info_array, int gbstep)
{
    const int batchid = blockIdx.x;
    extern __shared__ float shared_data[];

    int gboff = paneloffset+step;
    magma_int_t *ipiv           = ipiv_array[batchid] + ai;
    float *A_start = dA_array[batchid] + aj * lda + ai;
    float *A0j     = &(A_start[paneloffset + (paneloffset+step) * lda]);
    float *A00     = &(A_start[paneloffset + paneloffset * lda]);

    float *shared_A = shared_data;
    __shared__ float  shared_x[zamax];
    __shared__ int     shared_idx[zamax];
    __shared__ float alpha;
    int tid = threadIdx.x;

    // checkinfo to avoid computation of the singular matrix
    if (info_array[batchid] != 0 ) return;


    int nchunk = magma_ceildiv( m, MAX_NTHREADS );
    // read the current column from dev to shared memory
    for (int s=0; s < nchunk; s++)
    {
        if ( (tid + s * MAX_NTHREADS) < m ) shared_A[tid + s * MAX_NTHREADS] = A0j[tid + s * MAX_NTHREADS];
    }
    __syncthreads();

    // update this column
    if ( step > 0 ) {
        zupdate_device( m, step, A00, lda, shared_A, 1);
        __syncthreads();
    }

    // if ( tid < (m-step) ) // DO NO TPUT THE IF CONDITION HERE SINCE isamax_devfunc HAS __syncthreads INSIDE.
    // So let all htreads call this routine it will handle correctly based on the size
    // note that isamax need only 128 threads, s
    isamax_devfunc(m-step, shared_A+step, 1, shared_x, shared_idx);
    if (tid == 0) {
        ipiv[gboff]  = shared_idx[0] + gboff + 1; // Fortran Indexing
        alpha = shared_A[shared_idx[0]+step];
        //printf("@ step %d ipiv=%d where gboff=%d  shared_idx %d alpha %5.3f\n",step,ipiv[gboff],gboff,shared_idx[0],alpha);
        if (shared_x[0] == MAGMA_D_ZERO) {
            info_array[batchid] = shared_idx[0] + gboff + gbstep + 1;
        }
    }
    __syncthreads();
    if (shared_x[0] == MAGMA_D_ZERO) return;
    __syncthreads();

    // DO NO PUT THE IF CONDITION HERE SINCE isamax_devfunc HAS __syncthreads INSIDE.
    sscal5_device( m-step, shared_A+step, alpha);

    // put back the pivot that has been scaled with itself menaing =1
    if (tid == 0)  shared_A[shared_idx[0] + step] = alpha;
    __syncthreads();

    // write back from shared to dev memory
    for (int s=0; s < nchunk; s++)
    {
        if ( (tid + s * MAX_NTHREADS) < m )
        {
            A0j[tid + s * MAX_NTHREADS] = shared_A[tid + s * MAX_NTHREADS];
            //printf("@ step %d tid %d updating A=x*alpha after A= %5.3f\n",step,tid,shared_A[tid]);
        }
    }
    __syncthreads();
}


/******************************************************************************/
extern "C"
magma_int_t magma_scomputecolumn_batched( magma_int_t m, magma_int_t paneloffset, magma_int_t step,
                                          float **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t lda,
                                          magma_int_t **ipiv_array,
                                          magma_int_t *info_array, magma_int_t gbstep,
                                          magma_int_t batchCount, magma_queue_t queue)
{
    /*
    Specialized kernel which merged sscal and sger the two kernels
    1) sscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a sger Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( m == 0) return 0;

    size_t all_shmem_size = zamax*(sizeof(float)+sizeof(int)) + (m+2)*sizeof(float);
    if ( all_shmem_size >  (MAX_SHARED_ALLOWED*1024) ) // limit the shared memory to 44K leaving 4K for extra
    {
        fprintf( stderr, "%s error out of shared memory\n", __func__ );
        return -20;
    }

    size_t shared_size = sizeof(float)*m;
    dim3 grid(batchCount, 1, 1);
    dim3 threads(min(m, MAX_NTHREADS), 1, 1);

    zcomputecolumn_kernel_shared_batched
    <<< grid, threads, shared_size, queue->hip_stream() >>>
    (m, paneloffset, step, dA_array, ai, aj, lda, ipiv_array, info_array, gbstep);

    return 0;
}

/******************************************************************************/
template<int WIDTH>
__global__ void
sgetf2_fused_batched_kernel( int m,
                           float** dA_array, int ai, int aj, int ldda,
                           magma_int_t** dipiv_array, magma_int_t* info_array, int batchCount)
{
    // different indices per branch
    const int batchid = blockIdx.x * blockDim.y + threadIdx.y;
    //const int batchid = blockIdx.z * blockDim.y + threadIdx.y;

    extern __shared__ float zdata[];

    float* swork = (float*)zdata;
     if(batchid >= batchCount)return;
     sgetf2_fused_device<WIDTH>(
             m, dA_array[batchid] + aj * ldda + ai, ldda,
             dipiv_array[batchid] + ai,
             swork, &info_array[batchid], aj);
}


/***************************************************************************//**
    Purpose
    -------
    magma_sgetf2_reg_batched computes an LU factorization of a general M-by-N matrix A
    using partial pivoting with row interchanges. This routine is used for batch LU panel
    factorization, and has specific assumption about the value of N

    The factorization has the form
        A = P * L * U
    where P is a permutation matrix, L is lower triangular with unit
    diagonal elements (lower trapezoidal if m > n), and U is upper
    triangular (upper trapezoidal if m < n).

    This is a right-looking unblocked version of the algorithm. The routine is a batched
    version that factors batchCount M-by-N matrices in parallel.

    This version load an entire matrix (m*n) into registers and factorize it with pivoting
    and copy back to GPU device memory.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of each matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of each matrix A.  ib >= 0.

    @param[in,out]
    dA_array    Array of pointers, dimension (batchCount).
            Each is a REAL array on the GPU, dimension (LDDA,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ai      INTEGER
            Row offset for A.

    @param[in]
    aj      INTEGER
            Column offset for A.

    @param[in]
    ldda    INTEGER
            The leading dimension of each array A.  LDDA >= max(1,M).

    @param[out]
    dipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getf2_batched
*******************************************************************************/
extern "C" magma_int_t
magma_sgetf2_fused_batched(
    magma_int_t m, magma_int_t n,
    float **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t ldda,
    magma_int_t **dipiv_array,
    magma_int_t *info_array, magma_int_t batchCount,
    magma_queue_t queue)
{
    if(m < 0 || m > SGETF2_FUSED_BATCHED_MAX_ROWS) {
        fprintf( stderr, "%s: m = %4lld not supported, must be between 0 and %4lld\n",
                 __func__, (long long) m, (long long) SGETF2_FUSED_BATCHED_MAX_ROWS);
        return -1;
    }
    else if(n < 0 || n > 32){
        fprintf( stderr, "%s: n = %4lld not supported, must be between 0 and %4lld\n",
                 __func__, (long long) m, (long long) 32);
        return -2;
    }
    magma_int_t ntcol = (m > 32)? 1 : (2 * (32/m));

    magma_int_t shared_size = 0;
    shared_size += n * sizeof(float);
    shared_size += m * sizeof(float);
    shared_size += m * sizeof(int);    // not magma_int_t
    shared_size += n * sizeof(int);    // not magma_int_t
    shared_size *= ntcol;

    dim3 grid(magma_ceildiv(batchCount,ntcol), 1, 1);
    dim3 threads(m, ntcol, 1);

    switch(n)
    {
        case  1: sgetf2_fused_batched_kernel< 1><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  2: sgetf2_fused_batched_kernel< 2><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  3: sgetf2_fused_batched_kernel< 3><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  4: sgetf2_fused_batched_kernel< 4><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  5: sgetf2_fused_batched_kernel< 5><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  6: sgetf2_fused_batched_kernel< 6><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  7: sgetf2_fused_batched_kernel< 7><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  8: sgetf2_fused_batched_kernel< 8><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  9: sgetf2_fused_batched_kernel< 9><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 10: sgetf2_fused_batched_kernel<10><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 11: sgetf2_fused_batched_kernel<11><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 12: sgetf2_fused_batched_kernel<12><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 13: sgetf2_fused_batched_kernel<13><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 14: sgetf2_fused_batched_kernel<14><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 15: sgetf2_fused_batched_kernel<15><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 16: sgetf2_fused_batched_kernel<16><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 17: sgetf2_fused_batched_kernel<17><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 18: sgetf2_fused_batched_kernel<18><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 19: sgetf2_fused_batched_kernel<19><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 20: sgetf2_fused_batched_kernel<20><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 21: sgetf2_fused_batched_kernel<21><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 22: sgetf2_fused_batched_kernel<22><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 23: sgetf2_fused_batched_kernel<23><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 24: sgetf2_fused_batched_kernel<24><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 25: sgetf2_fused_batched_kernel<25><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 26: sgetf2_fused_batched_kernel<26><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 27: sgetf2_fused_batched_kernel<27><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 28: sgetf2_fused_batched_kernel<28><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 29: sgetf2_fused_batched_kernel<29><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 30: sgetf2_fused_batched_kernel<30><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 31: sgetf2_fused_batched_kernel<31><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 32: sgetf2_fused_batched_kernel<32><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        default: fprintf( stderr, "%s: n = %4lld is not supported \n", __func__, (long long) n);
    }
    return 0;
}
